#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>

using namespace std::chrono;

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void restVectorsInto(float *result, float *a, float *b, int N)
{
  // Process multiple elements per thread
  const int elementsPerThread = 4;
  int index = (threadIdx.x + blockIdx.x * blockDim.x) * elementsPerThread;
  
  #pragma unroll
  for(int i = 0; i < elementsPerThread && index + i < N; i++)
  {
    int idx = index + i;
    if (idx < N) {
      float diff = a[idx] - b[idx];
      result[idx] = diff * diff;
    }
  }
}

// Single-kernel reduction optimized for small arrays (N <= 1024)
__global__ void smallArrayReductionKernel(float *input, float *output, int N) {
    extern __shared__ float sdata[];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = tid;
    
    // Load input into shared memory
    sdata[tid] = (i < N) ? input[i] : 0;
    __syncthreads();
    
    // Unrolled reduction for better performance
    if (N >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (N >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (N >= 128) { if (tid < 64)  { sdata[tid] += sdata[tid + 64]; }  __syncthreads(); }
    
    // Last warp reduction (no sync needed)
    if (tid < 32) {
        volatile float* smem = sdata;
        if (N >= 64) smem[tid] += smem[tid + 32];
        if (N >= 32) smem[tid] += smem[tid + 16];
        if (N >= 16) smem[tid] += smem[tid + 8];
        if (N >= 8)  smem[tid] += smem[tid + 4];
        if (N >= 4)  smem[tid] += smem[tid + 2];
        if (N >= 2)  smem[tid] += smem[tid + 1];
    }
    
    if (tid == 0) output[0] = sdata[0];
}

// Simple wrapper for parallel reduction
float parallelReduceSum(float* d_input, int N, hipStream_t& stream) {
    float final_sum;
    float *d_output;
    hipMalloc(&d_output, sizeof(float));
    
    // Round up to nearest warp size (32)
    int threadsNeeded = (N + 31) / 32 * 32;
    
    // Launch single kernel for reduction
    smallArrayReductionKernel<<<1, threadsNeeded, threadsNeeded * sizeof(float), stream>>>(
        d_input, d_output, N
    );
    
    // Copy result back to host
    hipMemcpyAsync(&final_sum, d_output, sizeof(float), 
                   hipMemcpyDeviceToHost, stream);
    
    hipStreamSynchronize(stream);
    hipFree(d_output);
    
    return final_sum;
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 128;
  size_t size = N * sizeof(float);

  // Create CUDA stream for async operations
  hipStream_t stream;
  hipStreamCreate(&stream);

  // Allocate memory
  float *a, *b, *c;
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  // Prefetch data to GPU
  hipMemPrefetchAsync(a, size, deviceId, stream);
  hipMemPrefetchAsync(b, size, deviceId, stream);
  hipMemPrefetchAsync(c, size, deviceId, stream);

  // Initialize data
  initWith(2, a, N);
  initWith(1, b, N);
  initWith(0, c, N);

  // Optimize grid and block dimensions
  int threadsPerBlock = 256;
  int blocksPerSM = 32;
  int numberOfBlocks = numberOfSMs * blocksPerSM;
  
  // Ensure we have enough threads to cover all elements
  numberOfBlocks = (N + threadsPerBlock * 4 - 1) / (threadsPerBlock * 4);

  // Start timing after setup
  hipStreamSynchronize(stream);
  clock_t start = clock();

  // Launch kernel
  restVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream>>>(c, a, b, N);

  // Check for errors
  hipError_t addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  // Compute final sum using parallel reduction
  float sum = parallelReduceSum(c, N, stream);
  sum = sqrt(sum);

  clock_t end = clock();
  double duration = ((double)(end - start)) / CLOCKS_PER_SEC * 1000000;
  printf("Time taken: %.2f microseconds\n", duration);
  printf("Result: %.2f\n", sum);

  // Cleanup
  hipStreamDestroy(stream);
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
