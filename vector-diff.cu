#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>

using namespace std::chrono;

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__
void restVectorsInto(float *result, float *a, float *b, int N)
{
  // Process multiple elements per thread
  const int elementsPerThread = 4;
  int index = (threadIdx.x + blockIdx.x * blockDim.x) * elementsPerThread;
  
  #pragma unroll
  for(int i = 0; i < elementsPerThread && index + i < N; i++)
  {
    int idx = index + i;
    if (idx < N) {
      float diff = a[idx] - b[idx];
      result[idx] = diff * diff;
    }
  }
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 128;
  size_t size = N * sizeof(float);

  // Create CUDA stream for async operations
  hipStream_t stream;
  hipStreamCreate(&stream);

  // Allocate memory
  float *a, *b, *c;
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  // Prefetch data to GPU
  hipMemPrefetchAsync(a, size, deviceId, stream);
  hipMemPrefetchAsync(b, size, deviceId, stream);
  hipMemPrefetchAsync(c, size, deviceId, stream);

  // Initialize data
  initWith(2, a, N);
  initWith(1, b, N);
  initWith(0, c, N);

  // Optimize grid and block dimensions
  int threadsPerBlock = 256;
  int blocksPerSM = 32;
  int numberOfBlocks = numberOfSMs * blocksPerSM;
  
  // Ensure we have enough threads to cover all elements
  numberOfBlocks = (N + threadsPerBlock * 4 - 1) / (threadsPerBlock * 4);

  // Start timing after setup
  hipStreamSynchronize(stream);
  clock_t start = clock();

  // Launch kernel
  restVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream>>>(c, a, b, N);

  // Check for errors
  hipError_t addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  // Synchronize and get result
  hipStreamSynchronize(stream);
  
  // Calculate final result
  float sum = 0;
  for (int i = 0; i < N; i++) {
    sum += c[i];
  }
  sum = sqrt(sum);

  clock_t end = clock();
  double duration = ((double)(end - start)) / CLOCKS_PER_SEC * 1000000;
  printf("Time taken: %.2f microseconds\n", duration);
  printf("Result: %.2f\n", sum);

  // Cleanup
  hipStreamDestroy(stream);
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
