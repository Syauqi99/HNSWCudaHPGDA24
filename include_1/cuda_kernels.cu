#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "cuda_kernels.cuh"
#include <vector>
using namespace std;

namespace hnsw {

// CUDA kernel declarations
__device__ float euclidean_distance_cuda(const float* a, const float* b, int dim) {
    float sum = 0.0f;
    for (int i = 0; i < dim; i++) {
        float diff = a[i] - b[i];
        sum += diff * diff;
    }
    return sqrt(sum);
}

__global__ void batch_distance_calculation(const float* queries, 
                                        const float* dataset,
                                        float* distances,
                                        int n_queries,
                                        int n_points,
                                        int dim) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_queries * n_points) {
        int query_idx = idx / n_points;
        int point_idx = idx % n_points;
        
        distances[idx] = euclidean_distance_cuda(
            &queries[query_idx * dim],
            &dataset[point_idx * dim],
            dim
        );
    }
}

void cuda_batch_distance_calculation(const std::vector<float>& queries,
                                   const std::vector<float>& dataset,
                                   std::vector<float>& distances,
                                   int n_queries,
                                   int n_points,
                                   int dim) {
    // Allocate device memory
    float *d_queries, *d_dataset, *d_distances;
    hipMalloc(&d_queries, queries.size() * sizeof(float));
    hipMalloc(&d_dataset, dataset.size() * sizeof(float));
    hipMalloc(&d_distances, distances.size() * sizeof(float));

    // Copy data to device
    hipMemcpy(d_queries, queries.data(), queries.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dataset, dataset.data(), dataset.size() * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n_queries * n_points + threadsPerBlock - 1) / threadsPerBlock;
    batch_distance_calculation<<<blocksPerGrid, threadsPerBlock>>>(
        d_queries, d_dataset, d_distances, n_queries, n_points, dim);

    // Copy results back to host
    hipMemcpy(distances.data(), d_distances, distances.size() * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_queries);
    hipFree(d_dataset);
    hipFree(d_distances);
}

// Implementation of the template function
template <typename T>
float cuda_euclidean_distance(const vector<float>& p1, const vector<float>& p2) {
    int dim = p1.size();
    vector<float> result(1);

    // Allocate device memory
    float *d_vec1, *d_vec2, *d_result;
    hipMalloc(&d_vec1, dim * sizeof(float));
    hipMalloc(&d_vec2, dim * sizeof(float));
    hipMalloc(&d_result, sizeof(float));

    // Copy data to device
    hipMemcpy(d_vec1, p1.data(), dim * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, p2.data(), dim * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (dim + threadsPerBlock - 1) / threadsPerBlock;
    batch_distance_calculation<<<blocksPerGrid, threadsPerBlock>>>(
        d_vec1, d_vec2, d_result, 1, 1, dim);

    // Copy result back to host
    hipMemcpy(result.data(), d_result, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_vec1);
    hipFree(d_vec2);
    hipFree(d_result);

    return result[0];
}

// Explicit template instantiation
template float cuda_euclidean_distance<float>(const vector<float>&, const vector<float>&);

} // namespace hnsw 