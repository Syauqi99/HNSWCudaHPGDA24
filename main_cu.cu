#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <chrono>
#include <iostream>  // Added this header for std::cout

__global__ void cuda_euclidean_distance(float *a, float *b, float *result, int N) {
    // Declare shared memory array - visible to all threads in the same block
    // Size must be specified when kernel is launched
    extern __shared__ float shared_data[];

    // Calculate global thread ID and stride
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;  // Global thread ID
    int gridStride = blockDim.x * gridDim.x;               // Total number of threads
    int localId = threadIdx.x;                             // Local thread ID within the block

    // Initialize local sum for this thread
    float sum = 0.0f;

    // Each thread processes multiple elements with grid-stride loop
    // This allows handling arrays larger than total number of threads
    for (int i = threadId; i < N; i += gridStride) {
        float diff = a[i] - b[i];        // Calculate difference
        sum += diff * diff;              // Add squared difference to local sum
    }

    // Store this thread's sum in shared memory
    shared_data[localId] = sum;
    
    // Ensure all threads in block have written to shared memory
    __syncthreads();

    // Parallel reduction in shared memory
    // This loop reduces the partial sums in shared memory to a single sum per block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (localId < stride) {
            // Each thread adds a value from the second half to the first half
            shared_data[localId] += shared_data[localId + stride];
        }
        // Ensure all threads have finished reading shared memory before next iteration
        __syncthreads();
    }

    // Only thread 0 in each block writes the final result
    if (localId == 0) {
        // Atomically add this block's sum to the global result
        // atomicAdd is necessary because multiple blocks may write simultaneously
        atomicAdd(result, shared_data[0]);
    }
}

int main() {
  int N = 1000000;  // Size of arrays

  
  // Allocate host memory
  h_a = (float*)malloc(N * sizeof(float));
  h_b = (float*)malloc(N * sizeof(float));
  h_result = (float*)malloc(sizeof(float));

  // Initialize host arrays
  for (int i = 0; i < N; i++) {
      h_a[i] = static_cast<float>(i);
      h_b[i] = static_cast<float>(i + 1);
  }
  *h_result = 0.0f;



  // Start timing
  auto start = std::chrono::high_resolution_clock::now();
  // Host code to launch the kernel
  float *d_a, *d_b, *d_result;
  size_t size = N * sizeof(float);

  // Allocate device memory
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_result, sizeof(float));

  if (checkCuda(hipMallocManaged(&d_a, size)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipMallocManaged(&d_b, size)) != hipSuccess){
    exit(1);
  }

  if (checkCuda(hipMallocManaged(&d_c, size)) != hipSuccess){
    exit(1);
  }

  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&props, deviceId);

  threadsPerBlock = props.maxThreadsPerBlock;
  numberOfBlocks = props.multiProcessorCount * 2;

  hipMemPrefetchAsync(d_a, size, deviceId);
  hipMemPrefetchAsync(d_b, size, deviceId);

  // Initialize result to 0
  hipMemset(d_result, 0, sizeof(float));

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  // Launch kernel with shared memory size = blockSize * sizeof(float)
  cuda_euclidean_distance<<<numberOfBlocks, threadsPerBlock>>>(
      d_a, d_b, d_result, N
  );

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  // End timing
  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();

  std::cout << "Time taken: " << duration << " microseconds" << std::endl;

  // Free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_result);
  hipMemPrefetchAsync(d_result, sizeof(float), hipCpuDeviceId); // Prefetch c to CPU

  return 0;
}
